#include "csr_utils.h"
#include <assert.h>
#include <hip/hip_runtime.h>
#include <hipsparse.h>
#include <stdio.h>
#include <stdlib.h>

#define CHECK_CUDA(func)                                                                           \
    {                                                                                              \
        hipError_t status = (func);                                                               \
        if (status != hipSuccess)                                                                 \
        {                                                                                          \
            printf("CUDA API failed at line %d with error: %s (%d)\n", __LINE__,                   \
                   hipGetErrorString(status), status);                                            \
            goto cleanup;                                                                          \
        }                                                                                          \
    }

#define CHECK_CUSPARSE(func)                                                                       \
    {                                                                                              \
        hipsparseStatus_t status = (func);                                                          \
        if (status != HIPSPARSE_STATUS_SUCCESS)                                                     \
        {                                                                                          \
            printf("CUSPARSE API failed at line %d with error: %s (%d)\n", __LINE__,               \
                   hipsparseGetErrorString(status), status);                                        \
            goto cleanup;                                                                          \
        }                                                                                          \
    }

extern "C"
{
    CSRMatrix cusparse_compute_AAT(int m, int n, int nnz, const int *csr_row_ptr,
                                   const int *csr_col_idx, const double *csr_val)
    {
        double alpha = 1.0;
        double beta = 0.0;
        hipsparseOperation_t opA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipsparseOperation_t opB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
        hipDataType computeType = HIP_R_64F;
        hipDataType valueType = HIP_R_64F;

        // Allocate and copy input matrix A to device
        int *d_row_ptr_A, *d_col_idx_A;
        double *d_val_A;
        int *d_row_ptr_AT, *d_col_idx_AT;
        double *d_val_AT;
        int *d_row_ptr_AAT, *d_col_idx_AAT;
        double *d_val_AAT;
        // CHECK_CUDA(hipMalloc((void **)&d_row_ptr_A, (m + 1) * sizeof(int)))
        // CHECK_CUDA(hipMalloc((void **)&d_col_idx_A, nnz * sizeof(int)))
        // CHECK_CUDA(hipMalloc((void **)&d_val_A, nnz * sizeof(double)))
        // CHECK_CUDA(hipMalloc((void **)&d_row_ptr_AT, (m + 1) * sizeof(int)))
        // CHECK_CUDA(hipMalloc((void **)&d_col_idx_AT, nnz * sizeof(int)))
        // CHECK_CUDA(hipMalloc((void **)&d_val_AT, nnz * sizeof(double)))
        hipMalloc((void **)&d_row_ptr_A, (m + 1) * sizeof(int));
        hipMalloc((void **)&d_col_idx_A, nnz * sizeof(int));
        hipMalloc((void **)&d_val_A, nnz * sizeof(double));
        hipMalloc((void **)&d_row_ptr_AT, (m + 1) * sizeof(int));
        hipMalloc((void **)&d_col_idx_AT, nnz * sizeof(int));
        hipMalloc((void **)&d_val_AT, nnz * sizeof(double));
        // column indices and values of AA^T are allocated later
        hipMalloc((void **)&d_row_ptr_AAT, (m + 1) * sizeof(int));
        hipMemcpy(d_row_ptr_A, csr_row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_col_idx_A, csr_col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_val_A, csr_val, nnz * sizeof(double), hipMemcpyHostToDevice);
        printf("d_val_AT: %p, d_col_idx_AT: %p, d_row_ptr_AT: %p\n", d_val_AT, d_col_idx_AT,
               d_row_ptr_AT);

        // -------------------------------------------------------------------------------------//

        // Create matrix descriptors
        hipsparseHandle_t handle = NULL;
        hipsparseSpMatDescr_t matA, matAT, matAAT;
        void *dBuffer1 = NULL, *dBuffer2 = NULL, *dBufferAT = NULL;
        size_t bufferSize1 = 0, bufferSize2 = 0, bufferSizeAT = 0;
        hipsparseCreate(&handle);

        hipsparseCsr2cscEx2_bufferSize(handle, m, n, nnz, d_val_A, d_row_ptr_A, d_col_idx_A,
                                      d_val_AT, d_col_idx_AT, d_row_ptr_AT, HIP_R_64F,
                                      HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO,
                                      HIPSPARSE_CSR2CSC_ALG1, &bufferSizeAT);
        printf("bufferSizeAT = %zu\n", bufferSizeAT);

        hipMalloc(&dBufferAT, bufferSizeAT);
        hipsparseCsr2cscEx2(handle, m, n, nnz, d_val_A, d_row_ptr_A, d_col_idx_A, d_val_AT,
                           d_col_idx_AT, d_row_ptr_AT, HIP_R_64F, HIPSPARSE_ACTION_NUMERIC,
                           HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, dBufferAT);

        hipsparseCreateCsr(&matA, m, n, nnz, d_row_ptr_A, d_col_idx_A, d_val_A, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, valueType);
        hipsparseCreateCsr(&matAT, n, m, nnz, d_row_ptr_AT, d_col_idx_AT, d_val_AT,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO,
                          valueType);
        hipsparseCreateCsr(&matAAT, m, m, 0, d_row_ptr_AAT, NULL, NULL, HIPSPARSE_INDEX_32I,
                          HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, valueType);

        // -------------------------------------------------------------------------------------//
        // Create SpGEMM descriptor
        hipsparseSpGEMMDescr_t spgemmDesc;
        hipsparseSpGEMM_createDescr(&spgemmDesc);

        // ask bufferSize1 bytes for external memory
        hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA, matAT, &beta, matAAT,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                      &bufferSize1, NULL);
        hipMalloc(&dBuffer1, bufferSize1);

        // Inspect matrix A and A^T to understand the memory requirements
        hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA, matAT, &beta, matAAT,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                      &bufferSize1, dBuffer1);

        // ask bufferSize2 bytes for external memory
        hipsparseSpGEMM_workEstimation(handle, opA, opB, &alpha, matA, matAT, &beta, matAAT,
                                      computeType, HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc,
                                      &bufferSize2, NULL);
        hipMalloc(&dBuffer2, bufferSize2);

        // compute the intermediate product of A * A^T
        hipsparseSpGEMM_compute(handle, opA, opB, &alpha, matA, matAT, &beta, matAAT, computeType,
                               HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc, &bufferSize2, dBuffer2);

        // get matrix AAT non-zero entries AAT_nnz1
        int64_t C_num_rows1, C_num_cols1, AAT_nnz1;
        hipsparseSpMatGetSize(matAAT, &C_num_rows1, &C_num_cols1, &AAT_nnz1);

        // allocate matrix AAT
        hipMalloc((void **)&d_col_idx_AAT, AAT_nnz1 * sizeof(int));
        hipMalloc((void **)&d_val_AAT, AAT_nnz1 * sizeof(double));

        // Update AAT with the new poitners
        hipsparseCsrSetPointers(matAAT, d_row_ptr_AAT, d_col_idx_AAT, d_val_AAT);

        // -------------------------------------------------------------------------------------//
        hipsparseSpGEMM_copy(handle, opA, opB, &alpha, matA, matAT, &beta, matAAT, computeType,
                            HIPSPARSE_SPGEMM_DEFAULT, spgemmDesc);
        // -------------------------------------------------------------------------------------//
        hipsparseDestroySpMat(matA);
        hipsparseDestroySpMat(matAT);
        hipsparseDestroySpMat(matAAT);
        hipsparseSpGEMM_destroyDescr(spgemmDesc);
        hipsparseDestroy(handle);
        // -------------------------------------------------------------------------------------//
        int *h_row_ptr_AAT = (int *)malloc((m + 1) * sizeof(int));
        int *h_col_idx_AAT = (int *)malloc(AAT_nnz1 * sizeof(int));
        double *h_val_AAT = (double *)malloc(AAT_nnz1 * sizeof(double));
        hipMemcpy(h_row_ptr_AAT, d_row_ptr_AAT, (m + 1) * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_col_idx_AAT, d_col_idx_AAT, AAT_nnz1 * sizeof(int), hipMemcpyDeviceToHost);
        hipMemcpy(h_val_AAT, d_val_AAT, AAT_nnz1 * sizeof(double), hipMemcpyDeviceToHost);
        // -------------------------------------------------------------------------------------//
        hipFree(dBuffer1);
        hipFree(dBuffer2);
        hipFree(dBufferAT);
        hipFree(d_row_ptr_A);
        hipFree(d_col_idx_A);
        hipFree(d_val_A);
        hipFree(d_row_ptr_AT);
        hipFree(d_col_idx_AT);
        hipFree(d_val_AT);
        hipFree(d_row_ptr_AAT);
        hipFree(d_col_idx_AAT);
        hipFree(d_val_AAT);
        CSRMatrix result = {h_row_ptr_AAT, h_col_idx_AAT, h_val_AAT, (int)AAT_nnz1};
        //     goto done;

        // cleanup:
        //     if (dBuffer1)
        //         hipFree(dBuffer1);
        //     if (dBuffer2)
        //         hipFree(dBuffer2);
        //     if (dBufferAT)
        //         hipFree(dBufferAT);
        //     if (d_row_ptr_A)
        //         hipFree(d_row_ptr_A);
        //     if (d_col_idx_A)
        //         hipFree(d_col_idx_A);
        //     if (d_val_A)
        //         hipFree(d_val_A);
        //     if (d_row_ptr_AT)
        //         hipFree(d_row_ptr_AT);
        //     if (d_col_idx_AT)
        //         hipFree(d_col_idx_AT);
        //     if (d_val_AT)
        //         hipFree(d_val_AT);
        //     if (d_row_ptr_AAT)
        //         hipFree(d_row_ptr_AAT);
        //     if (d_col_idx_AAT)
        //         hipFree(d_col_idx_AAT);
        //     if (d_val_AAT)
        //         hipFree(d_val_AAT);
        //     if (matA)
        //         hipsparseDestroySpMat(matA);
        //     if (matAT)
        //         hipsparseDestroySpMat(matAT);
        //     if (matAAT)
        //         hipsparseDestroySpMat(matAAT);
        //     if (spgemmDesc)
        //         hipsparseSpGEMM_destroyDescr(spgemmDesc);
        //     if (handle)
        //         hipsparseDestroy(handle);
        //     CSRMatrix empty = {NULL, NULL, NULL, 0};
        //     return empty;
        //     //
        //     -------------------------------------------------------------------------------------//

        // done:
        return result;
    }
}
