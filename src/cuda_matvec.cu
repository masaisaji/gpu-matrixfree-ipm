#include "csr_utils.h"
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hipsparse.h>
#include <stdio.h>

extern "C" void init_cuda()
{
    // Optional: reset device (only use for full reset, not usually necessary)
    // hipDeviceReset();

    // Force context initialization
    hipError_t cerr = hipFree(0);
    if (cerr != hipSuccess)
    {
        fprintf(stderr, "hipFree(0) failed: %s\n", hipGetErrorString(cerr));
        return;
    }

    hipsparseHandle_t handle;
    hipsparseStatus_t status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        fprintf(stderr, "cuSPARSE initialization failed! Status = %d\n", status);
        return;
    }

    hipsparseDestroy(handle);
}

extern "C" void cuda_matvec_csr(const CSRMatrix *A, const double *h_x, double *Ax)
{

    //--------------------------------------------------------------------------
    hipEvent_t start, stop;
    float elapsedTime;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    // Unpack CSRMatrix A
    int m = A->rows;
    int n = A->cols;
    int nnz = A->nnz;
    const double *csr_values = A->val;
    const int *csr_row_ptr = A->row_ptr;
    const int *csr_col_idx = A->col_idx;

    // Allocate device memory
    double *d_vals, *d_x, *d_y;
    int *d_row_ptr, *d_col_idx;
    hipMalloc(&d_vals, nnz * sizeof(double));
    hipMalloc(&d_row_ptr, (m + 1) * sizeof(int));
    hipMalloc(&d_col_idx, nnz * sizeof(int));
    hipMalloc(&d_x, n * sizeof(double));
    hipMalloc(&d_y, m * sizeof(double));

    // Copy data to device
    hipMemcpy(d_vals, csr_values, nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_row_ptr, csr_row_ptr, (m + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, csr_col_idx, nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_x, h_x, n * sizeof(double), hipMemcpyHostToDevice);

    //--------------------------------------------------------------------------
    // CUSPARSE APIs
    double alpha = 1.0, beta = 0.0;

    hipsparseHandle_t handle;
    hipsparseStatus_t status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS)
    {
        fprintf(stderr, "hipsparseCreate failed: %d\n", status);
        return;
    }

    hipsparseSpMatDescr_t matA;
    hipsparseDnVecDescr_t vecX, vecY;

    // Create matrix A in CSR format
    hipsparseCreateCsr(&matA, m, n, nnz, d_row_ptr, d_col_idx, d_vals, HIPSPARSE_INDEX_32I,
                      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);
    hipsparseCreateDnVec(&vecX, n, d_x, HIP_R_64F);
    hipsparseCreateDnVec(&vecY, m, d_y, HIP_R_64F);

    //--------------------------------------------------------------------------
    hipEventRecord(start, 0);
    //--------------------------------------------------------------------------

    // allocate buffter
    size_t bufferSize = 0;
    void *dBuffer = NULL;

    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta,
                            vecY, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);

    // execute preprocess (optional)
    hipsparseSpMV_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta,
                            vecY, HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    // excecute SpMV
    hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, &alpha, matA, vecX, &beta, vecY,
                 HIP_R_64F, HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

    //--------------------------------------------------------------------------
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    // printf("Time elapsed: %.6f s\n", elapsedTime / 1000.0f);
    //--------------------------------------------------------------------------

    // Cleanup
    hipsparseDestroySpMat(matA);
    hipsparseDestroyDnVec(vecX);
    hipsparseDestroyDnVec(vecY);
    hipsparseDestroy(handle);

    // copy result back to host
    hipMemcpy(Ax, d_y, m * sizeof(double), hipMemcpyDeviceToHost);

    // Cleanup
    hipFree(d_vals);
    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_x);
    hipFree(d_y);
    hipFree(dBuffer);
    //--------------------------------------------------------------------------
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //--------------------------------------------------------------------------
}

// Allocates and uploads A (dense matrix), x (vector), and allocates y (vector)
extern "C" void alloc_and_upload_dense_matvec(const double *A_host, const double *x_host,
                                              int A_rows, int A_cols, double **d_A, double **d_x,
                                              double **d_y)
{
    {
        size_t A_size = A_rows * A_cols * sizeof(double);
        size_t x_size = A_cols * sizeof(double); // assuming A*x, so x has dim A_cols
        size_t y_size = A_rows * sizeof(double); // y has dim A_rows

        hipMalloc((void **)d_A, A_size);
        hipMalloc((void **)d_x, x_size);
        hipMalloc((void **)d_y, y_size);

        hipMemcpy(*d_A, A_host, A_size, hipMemcpyHostToDevice);
        hipMemcpy(*d_x, x_host, x_size, hipMemcpyHostToDevice);
        hipMemset(*d_y, 0, y_size); // optionally zero y
    }
}

// Dense matrix-vector multiplication using cuBLAS
extern "C" void cuda_matvec_dense(const double *d_A, const double *d_x, double *d_y, double *h_y,
                                  int A_rows, int A_cols, bool transpose)
{
    {
        hipblasHandle_t handle;
        hipblasCreate(&handle);

        const double alpha = 1.0;
        const double beta = 0.0;
        hipblasOperation_t op = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        hipblasDgemv(handle, op, A_rows, A_cols, &alpha, d_A, A_rows, d_x, 1, &beta, d_y, 1);

        hipblasDestroy(handle);
        if (h_y)
        {
            size_t y_size = (transpose ? A_cols : A_rows) * sizeof(double);
            hipMemcpy(h_y, d_y, y_size, hipMemcpyDeviceToHost);
        }
    }
}

extern "C" void free_cuda_matvec_memory(double *d_A, double *d_x, double *d_y)
{
    {
        if (d_A)
            hipFree(d_A);
        if (d_x)
            hipFree(d_x);
        if (d_y)
            hipFree(d_y);
    }
}

extern "C" void cuda_matvec_dense_handle(hipblasHandle_t handle, const double *d_A,
                                         const double *d_x, double *d_y, double *h_y, int A_rows,
                                         int A_cols, bool transpose)
{
    {
        const double alpha = 1.0;
        const double beta = 0.0;
        hipblasOperation_t op = transpose ? HIPBLAS_OP_T : HIPBLAS_OP_N;

        hipblasDgemv(handle, op, A_rows, A_cols, &alpha, d_A, A_rows, d_x, 1, &beta, d_y, 1);

        if (h_y)
        {
            size_t y_size = (transpose ? A_cols : A_rows) * sizeof(double);
            hipMemcpy(h_y, d_y, y_size, hipMemcpyDeviceToHost);
        }
    }
}
